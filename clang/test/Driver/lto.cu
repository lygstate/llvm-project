
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// -flto causes a switch to llvm-bc object files.
// RUN: %clangxx -nocudainc -nocudalib -ccc-print-phases -c %s -flto 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-ACTIONS < %t %s
//
// CHECK-COMPILE-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILE-ACTIONS-NOT: lto-bc
// CHECK-COMPILE-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)

// RUN: %clangxx -nocudainc -nocudalib -ccc-print-phases %s -flto 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-ACTIONS < %t %s
//
// CHECK-COMPILELINK-ACTIONS: 0: input, "{{.*}}lto.cu", cuda, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 1: preprocessor, {0}, cuda-cpp-output
// CHECK-COMPILELINK-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 3: input, "{{.*}}lto.cu", cuda, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 5: compiler, {4}, ir, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 6: backend, {5}, assembler, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 7: assembler, {6}, object, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 8: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_20)" {7}, object
// CHECK-COMPILELINK-ACTIONS: 9: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_20)" {6}, assembler
// CHECK-COMPILELINK-ACTIONS: 10: linker, {8, 9}, cuda-fatbin, (device-cuda)
// CHECK-COMPILELINK-ACTIONS: 11: offload, "host-cuda {{.*}}" {2}, "device-cuda{{.*}}" {10}, ir
// CHECK-COMPILELINK-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 13: linker, {12}, image, (host-cuda)

// llvm-bc and llvm-ll outputs need to match regular suffixes
// (unfortunately).
// RUN: %clangxx %s -nocudainc -nocudalib -flto -save-temps -### 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-SUFFIXES < %t %s
//
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[CPP:.*lto-host.*\.cui]]" "-x" "cuda" "{{.*}}lto.cu"
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[BC:.*lto-host.*\.bc]]" {{.*}}[[CPP]]"
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[OBJ:.*lto-host.*\.o]]" {{.*}}[[BC]]"
// CHECK-COMPILELINK-SUFFIXES: "{{.*}}a.{{(out|exe)}}" {{.*}}[[OBJ]]"

// RUN: %clangxx %s -nocudainc -nocudalib -flto -S -### 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-SUFFIXES < %t %s
//
// CHECK-COMPILE-SUFFIXES: "-o" "{{.*}}lto.s" "-x" "cuda" "{{.*}}lto.cu"

// RUN: not %clangxx -nocudainc -nocudalib %s -emit-llvm 2>&1 \
// RUN:    | FileCheck --check-prefix=LLVM-LINK %s
// LLVM-LINK: -emit-llvm cannot be used when linking

// -flto should cause link using gold plugin
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-LTO-ACTION < %t %s
//
// CHECK-LINK-LTO-ACTION: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"

// -flto=full should cause link using gold plugin
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto=full 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-FULL-ACTION < %t %s
//
// CHECK-LINK-FULL-ACTION: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"

// Check that subsequent -fno-lto takes precedence
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto=full -fno-lto 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-NOLTO-ACTION < %t %s
//
// CHECK-LINK-NOLTO-ACTION-NOT: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"

// -flto passes along an explicit debugger tuning argument.
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto -glldb 2> %t
// RUN: FileCheck -check-prefix=CHECK-TUNING-LLDB < %t %s
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto -g 2> %t
// RUN: FileCheck -check-prefix=CHECK-NO-TUNING < %t %s
//
// CHECK-TUNING-LLDB:   "-plugin-opt=-debugger-tune=lldb"
// CHECK-NO-TUNING-NOT: "-plugin-opt=-debugger-tune
